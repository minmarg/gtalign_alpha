#include "hip/hip_runtime.h"
/***************************************************************************
 *   Copyright (C) 2021-2023 Mindaugas Margelevicius                       *
 *   Institute of Biotechnology, Vilnius University                        *
 ***************************************************************************/

// All kernels in this module designed to process covariance and related 
// data following the application of DP and obtaining of matched positions

#include "libutil/cnsts.h"
#include "libutil/macros.h"
#include "libgenp/gproc/gproc.h"
#include "libgenp/gdats/PM2DVectorFields.h"

#include "libmycu/cucom/cucommon.h"
#include "libmycu/cucom/warpscan.cuh"
#include "libmycu/cuproc/cuprocconf.h"
#include "libmycu/culayout/cuconstant.cuh"
#include "libmycu/custages/stagecnsts.cuh"
#include "libmycu/custages/covariance.cuh"
#include "libmycu/custages/covariance_plus.cuh"
#include "libmycu/custages/covariance_refn.cuh"
#include "libmycu/custages/covariance_dp_refn.cuh"
#include "covariance_dp_scan.cuh"

// -------------------------------------------------------------------------
// CopyCCDataToWrkMem2_DPscan: copy cross-covariance matrix between the 
// query and reference structures to section 2 to enable efficient Kabsch 
// algorithm application for multiple structures simultaneously; 
// Version for alignments obtained as a result of the application of DP;
// NOTE: thread block is 2D and copies structures' data: from:
// NOTE: | struct i          | struct i+1        | ...
// NOTE: | field1,dield2,... | field1,dield2,... | ...
// NOTE: to 
// NOTE: | struct i | struct i+1 | ... | struct i | ... 
// NOTE: | field1   | field1     | ... | field2   | ...
// should be read;
// READNPOS, template parameter for checking whether nalnposs has changed;
// ndbCstrs, total number of reference structures in the chunk;
// maxnsteps, max number of steps (blockIdx.z) to perform for each 
// reference structure;
// NOTE: memory pointers should be aligned!
// wrkmemaux, auxiliary working memory;
// wrkmem, working memory, including the section of CC data (saved as 
// whole for each structure) to copy;
// wrkmem2, working memory, including the section of CC data to be written by 
// field;
// 
template<int READNPOS>
__global__ 
void CopyCCDataToWrkMem2_DPscan(
    const uint ndbCstrs,
    const uint maxnsteps,
    const float* __restrict__ wrkmemaux,
    const float* __restrict__ wrkmem,
    float* __restrict__ wrkmem2)
{
    constexpr int ndxproc = twmvEndOfCCDataExt;//index of processing flag in cache
    //cache for cross-covariance matrices and related data: 
    //bank conflicts resolved as long as innermost dim is odd
    __shared__ float ccmCache[CUS1_TBINITSP_CCMCOPY_N][twmvEndOfCCDataExt+1];
    //index of the first structure to start with (blockIdx.x, refn. serial number):
    int dbstrndx = blockIdx.x * CUS1_TBINITSP_CCMCOPY_N;
    int qryndx = blockIdx.y;//query index in the chunk
    int sfragfct = blockIdx.z;//fragment factor
    int absndx = dbstrndx + threadIdx.x;
    //int nalnposs = 0;

    if(absndx < ndbCstrs && 
       (threadIdx.y == tawmvConverged || threadIdx.y == tawmvNAlnPoss))
    {
        uint mloc = //tawmvConverged, convergence flag for a pair at sfragfct==0:
            (threadIdx.y == tawmvConverged)
            ? ((qryndx * maxnsteps + 0) * nTAuxWorkingMemoryVars + threadIdx.y) * ndbCstrs
            : ((qryndx * maxnsteps + sfragfct) * nTAuxWorkingMemoryVars + threadIdx.y) * ndbCstrs;
        ccmCache[threadIdx.x][threadIdx.y] = wrkmemaux[mloc + absndx/*dbstrndx*/];
    }

    __syncthreads();

    //write to smem #alignment positions
    if(absndx < ndbCstrs && threadIdx.y == 0)
    {
        ccmCache[threadIdx.x][ndxproc] = ccmCache[threadIdx.x][tawmvNAlnPoss];
        //any type of convergence applies
        if(ccmCache[threadIdx.x][tawmvConverged]) {
            //assign 0 #aligned positions so that no memory and 
            //computing operations are executed
            ccmCache[threadIdx.x][twmvNalnposs] = 0.0f;
            ccmCache[threadIdx.x][ndxproc] = 0.0f;
        }
    }

    __syncthreads();

    //cache data: iterative coalesced read
    for(int reldbndx = threadIdx.y; reldbndx < CUS1_TBINITSP_CCMCOPY_N; reldbndx += blockDim.y) {
        int absndxloc = dbstrndx + reldbndx;
        if(absndxloc < ndbCstrs && 
           threadIdx.x < twmvEndOfCCDataExt && 
           ccmCache[reldbndx][ndxproc])
        {
            //read only if not converged and not out of bounds
            uint mloc = ((qryndx * maxnsteps + sfragfct) * ndbCstrs + absndxloc) * nTWorkingMemoryVars;
            ccmCache[reldbndx][threadIdx.x] = wrkmem[mloc + threadIdx.x];
            if(READNPOS==READNPOS_READ &&
               threadIdx.x == twmvNalnposs && 
               ccmCache[reldbndx][twmvNalnposs] == ccmCache[reldbndx][ndxproc]) {
                //NOTE: if nalnposs equals maximum possible for given qrypos and rfnpos,
                //assign it to 0 so that the Kabsch algorithm is not applied to this 
                //particular query-reference pair:
                ccmCache[reldbndx][twmvNalnposs] = 0.0f;
                ccmCache[reldbndx][ndxproc] = 0.0f;
            }
        }
    }

    __syncthreads();

    //write data to gmem; coalesced write;
    //first write nalnposs 
    if(absndx < ndbCstrs && threadIdx.y == twmvNalnposs) {
        uint mloc = ((qryndx * maxnsteps + sfragfct) * nTWorkingMemoryVars + threadIdx.y) * ndbCstrs;
        wrkmem2[mloc + absndx] = ccmCache[threadIdx.x][threadIdx.y];
    }

    if(absndx < ndbCstrs && threadIdx.y < twmvEndOfCCData &&
       ccmCache[threadIdx.x][ndxproc]) {
        //write only if nalnposs >0;
        uint mloc = ((qryndx * maxnsteps + sfragfct) * nTWorkingMemoryVars + threadIdx.y) * ndbCstrs;
        wrkmem2[mloc + absndx] = ccmCache[threadIdx.x][threadIdx.y];
    }
}

// =========================================================================
// Instantiations:
// 
#define INSTANTIATE_CopyCCDataToWrkMem2_DPscan(tpREADNPOS) \
    template __global__ void CopyCCDataToWrkMem2_DPscan<tpREADNPOS>( \
        const uint ndbCstrs, const uint maxnsteps, \
        const float* __restrict__ wrkmemaux, \
        const float* __restrict__ wrkmem, \
        float* __restrict__ wrkmem2);

INSTANTIATE_CopyCCDataToWrkMem2_DPscan(READNPOS_NOREAD);
INSTANTIATE_CopyCCDataToWrkMem2_DPscan(READNPOS_READ);

// -------------------------------------------------------------------------
// CalcCCMatrices64_DPscan: calculate cross-covariance matrix between the 
// query and reference structures given an alignment between them;
// Version for alignments obtained as a result of the application of DP;
// NOTE: thread block is 1D and processes alignment fragment along structure
// positions;
// NOTE: Version for CUS1_TBINITSP_CCMCALC_XDIM==64!
// nqystrs, total number of queries;
// ndbCstrs, total number of reference structures in the chunk;
// ndbCposs, total number of reference positions in the chunk;
// maxnsteps, max number of steps (blockIdx.z) to perform for each 
// reference structure;
// NOTE: memory pointers should be aligned!
// wrkmemaux, auxiliary working memory;
// tmpdpalnpossbuffer, global address of aligned positions;
// wrkmem, working memory, including the section of CC data;
// 
__global__ 
void CalcCCMatrices64_DPscan(
    const uint nqystrs,
    const uint ndbCstrs,
    const uint ndbCposs,
    const uint dbxpad,
    const uint maxnsteps,
    const float* __restrict__ wrkmemaux,
    const float* __restrict__ tmpdpalnpossbuffer,
    float* __restrict__ wrkmem)
{
    // blockIdx.x is the block index of positions for query-reference pair;
    // blockIdx.y is the reference serial number;
    // blockIdx.z is the query serial number TIMES fragment factor;
    //cache for the cross-covarinace matrix and related data: 
    //no bank conflicts as long as twmvEndOfCCData is odd
    __shared__ float ccmCache[twmvEndOfCCData * CUS1_TBINITSP_CCMCALC_XDIM];
    //relative position index:
    const uint ndx0 = blockIdx.x * blockDim.x * CUS1_TBINITSP_CCMCALC_XFCT;
    const uint ndx = ndx0 + threadIdx.x;
    uint sfragfct = blockIdx.z / nqystrs;//fragment factor
    uint qryndx = blockIdx.z - sfragfct * nqystrs;//query serial number
    int qrylen, dbstrlen;//query and reference length
    //distances in positions to the beginnings of the query and reference structures:
    uint /*qrydst, */dbstrdst;
    const int qrypos = 0, rfnpos = 0;


    if(threadIdx.x == 0) {
        //NOTE: reuse ccmCache to read convergence flag at sfragfct==0:
        uint mloc = ((qryndx * maxnsteps + 0/*sfragfct*/) * nTAuxWorkingMemoryVars) * ndbCstrs;
        ccmCache[6] = wrkmemaux[mloc + tawmvConverged * ndbCstrs + blockIdx.y/*dbstrndx*/];
    }

    __syncthreads();

    if(ccmCache[6])
        //(NOTE:any type of convergence applies);
        //all threads in the block exit;
        return;

    //NOTE: no sync as long as ccmCache cell for convergence is not overwritten;


    //reuse ccmCache
    if(threadIdx.x == 0) {
        ((int*)ccmCache)[1] = GetDbStrDst(blockIdx.y);
        //((int*)ccmCache)[3] = GetQueryDst(qryndx);
    }
    if(threadIdx.x == tawmvNAlnPoss + 32) {
        //NOTE: reuse ccmCache to read #matched positions (tawmvNAlnPoss) written at sfragfct;
        //NOTE: use different warp; structure-specific-formatted data;
        uint mloc = ((qryndx * maxnsteps + sfragfct) * nTAuxWorkingMemoryVars) * ndbCstrs;
        ccmCache[threadIdx.x] = wrkmemaux[mloc + (threadIdx.x-32) * ndbCstrs + blockIdx.y/*dbstrndx*/];
    }

    __syncthreads();


    //NOTE: no bank conflict when two threads from the same warp access the same address;
    //blockDim.x includes several warps
    //TODO: integers in [0;16777216] can be exactly represented by float:
    //TODO: consider updating memory limits calculation or using int cache!
    dbstrdst = ((int*)ccmCache)[1];
    //qrydst = ((int*)ccmCache)[3];
    qrylen = dbstrlen = ccmCache[tawmvNAlnPoss+32];

    __syncthreads();


    if(qrylen <= qrypos + ndx0 || dbstrlen <= rfnpos + ndx0)
        //all threads in the block exit if thread 0 is out of bounds
        return;


    //qrylen == dbstrlen; reuse qrylen for original alignment length;

    //initialize cache:
    //(initialization in parts is more efficient wrt #registers)
    InitCCMCacheExtended<twmvEndOfCCData,0,twmvEndOfCCData>(ccmCache);


    const int dblen = ndbCposs + dbxpad;
    //offset to the beginning of the data along the y axis 
    // wrt query qryndx and maxnsteps: 
    const int yofff = (qryndx * maxnsteps + sfragfct) * dblen * nTDPAlignedPoss;

    #pragma unroll
    for(int i = 0; i < CUS1_TBINITSP_CCMCALC_XFCT; i++) {
        //manually unroll along data blocks by a factor of CUS1_TBINITSP_CCMCALC_XFCT
        if(!(/*qrypos + ndx + i * blockDim.x < qrylen &&*/
             rfnpos + ndx + i * blockDim.x < dbstrlen))
            break;
        //starting position in tmpdpalnpossbuffer for a pair:
        //NOTE: qrypos == rfnpos as well as lengths: use qrylen as the 
        //NOTE: original alignment length here;
        //NOTE: alignment written in reverse order:
        int pos = yofff + dbstrdst + qrylen-1 - (rfnpos + ndx + i * blockDim.x);
        UpdateCCMOneAlnPos_DPRefined(//no sync;
            pos, dblen,
            tmpdpalnpossbuffer,
            ccmCache
        );
    }

    //sync now:
    __syncthreads();

    //unroll by a factor 2
    if(threadIdx.x < (CUS1_TBINITSP_CCMCALC_XDIM>>1)) {
        #pragma unroll
        for(int i = 0; i < twmvEndOfCCData; i++)
            ccmCache[threadIdx.x * twmvEndOfCCData +i] +=
                ccmCache[(threadIdx.x + (CUS1_TBINITSP_CCMCALC_XDIM>>1)) * twmvEndOfCCData +i];
    }

    __syncthreads();

    //unroll warp
    if(threadIdx.x < 32) {
        #pragma unroll
        for(int i = 0; i < twmvEndOfCCData; i++) {
            float sum = ccmCache[threadIdx.x * twmvEndOfCCData + i];
            sum = mywarpreducesum(sum);
            //write to the first data slot of SMEM
            if(threadIdx.x == 0) ccmCache[i] = sum;
        }
    }

    //in case of twmvEndOfCCData gets larger than warpSize
    __syncthreads();

    uint mloc = ((qryndx * maxnsteps + sfragfct) * ndbCstrs + blockIdx.y/*dbstrndx*/) * nTWorkingMemoryVars;

    //only one block and its one thread writes nalnposs
    if(blockIdx.x == 0 && threadIdx.x == 0)
        wrkmem[mloc + twmvNalnposs] = dbstrlen;

    //add the result and write to global memory
    if(threadIdx.x < twmvEndOfCCData)
        atomicAdd(&wrkmem[mloc + threadIdx.x], ccmCache[threadIdx.x]);
}

// -------------------------------------------------------------------------
// FindD02ThresholdsCCM_DPscan: efficiently find distance thresholds 
// for the inclusion of aligned positions for CCM and rotation matrix 
// calculations during exhaustive application of DP;
// NOTE: thread block is 1D and processes alignment along structure
// positions;
// ndbCstrs, total number of reference structures in the chunk;
// ndbCposs, total number of db structure positions in the chunk;
// maxnsteps, max number of steps (blockIdx.z) to perform for each 
// reference structure;
// NOTE: memory pointers should be aligned!
// tmpdpdiagbuffers, temporary diagonal buffers filled with positional 
// scores;
// wrkmem, working memory, including the section of CC data;
// wrkmemaux, auxiliary working memory;
// 
template<int READCNST>
__global__
void FindD02ThresholdsCCM_DPscan(
    const uint ndbCstrs,
    const uint ndbCposs,
    const uint maxnsteps,
    const float* __restrict__ tmpdpdiagbuffers,
    const float* __restrict__ wrkmem,
    float* __restrict__ wrkmemaux)
{
    // blockIdx.x is the reference serial number;
    // blockIdx.y is the query serial number;
    // blockIdx.z is the fragment factor;
    //cache for minimum scores: 
    //no bank conflicts as long as inner-most dim is odd
    constexpr int smidim = 3;//top three min scores
    __shared__ float ccmCache[smidim * CUS1_TBINITSP_FINDD02_ITRD_XDIM];
    uint qryndx = blockIdx.y;//query serial number
    uint sfragfct = blockIdx.z;//fragment factor
    int qrylenorg, dbstrlenorg;//original query and reference lengths
    int qrylen, dbstrlen;//pseudo query and reference length, #matched positions
    //distances in positions to the beginnings of the query and reference structures:
    uint /*qrydst, */dbstrdst;
    constexpr int qrypos = 0;
    constexpr int rfnpos = 0;


    if(threadIdx.x == 0) {
        //NOTE: reuse ccmCache to read convergence flag at sfragfct==0:
        uint mloc = ((qryndx * maxnsteps + 0/*sfragfct*/) * nTAuxWorkingMemoryVars + tawmvConverged) * ndbCstrs;
        ccmCache[6] = wrkmemaux[mloc + blockIdx.x/*dbstrndx*/];
    }

    __syncthreads();

    if(ccmCache[6])
        //(NOTE:any type of convergence applies);
        //all threads in the block exit;
        return;

    //NOTE: no sync as long as ccmCache cell for convergence is not overwritten;


    //NOTE: pps2DLen and pps2DDist assumed to be adjacent: see PM2DVectorFields.h!
    //reuse ccmCache
    if(threadIdx.x < 2) {
        GetDbStrLenDst(blockIdx.x, (int*)ccmCache);
        //GetQueryLenDst(qryndx, (int*)ccmCache + 2);
        if(threadIdx.x == 0) ((int*)ccmCache)[2] = GetQueryLength(qryndx);
    }
    if(threadIdx.x == tawmvNAlnPoss + 32) {
        //NOTE: reuse ccmCache to read #matched positions (tawmvNAlnPoss) written at sfragfct;
        //NOTE: use different warp; structure-specific-formatted data;
        uint mloc = ((qryndx * maxnsteps + sfragfct) * nTAuxWorkingMemoryVars) * ndbCstrs;
        ccmCache[threadIdx.x] = wrkmemaux[mloc + (threadIdx.x-32) * ndbCstrs + blockIdx.x/*dbstrndx*/];
    }

    __syncthreads();

    //NOTE: no bank conflict when two threads from the same warp access the same address;
    dbstrlenorg = ((int*)ccmCache)[0]; dbstrdst = ((int*)ccmCache)[1];
    qrylenorg = ((int*)ccmCache)[2]; //qrydst = ((int*)ccmCache)[3];
    qrylen = dbstrlen = ccmCache[tawmvNAlnPoss+32];


    __syncthreads();


    if(READCNST == READCNST_CALC2) {
        if(threadIdx.x == 0) {
            //NOTE: reuse ccmCache[0] to contain twmvLastD02s. ccmCache[1] twmvNalnposs
            uint mloc = ((qryndx * maxnsteps + sfragfct) * ndbCstrs + blockIdx.x/*dbstrndx*/) * nTWorkingMemoryVars;
            ccmCache[1] = wrkmem[mloc + twmvNalnposs];
        }

        __syncthreads();

        int nalnposs = ccmCache[1];
        if(nalnposs == dbstrlen)
            //dbstrlen is #originally aligned positions;
            //all threads in the block exit;
            return;

        //cache will be overwritten below, sync
        __syncthreads();
    }


    //calculate the threshold over the original fragment
    //initialize cache
    #pragma unroll
    for(int i = 0; i < smidim; i++)
        ccmCache[threadIdx.x * smidim + i] = CP_LARGEDST;

    for(int rpos = threadIdx.x; qrypos + rpos < qrylen && rfnpos + rpos < dbstrlen;
        rpos += blockDim.x)
    {
        //manually unroll along alignment
        int mloc = (qryndx * maxnsteps + sfragfct) * ndbCposs;
        GetMinScoreOneAlnPos<smidim>(//no sync;
            mloc + dbstrdst + rpos,//position for scores
            tmpdpdiagbuffers,
            ccmCache
        );
    }

    //sync now:
    __syncthreads();

    //unroll until reaching warpSize; 
    for(int xdim = (CUS1_TBINITSP_FINDD02_ITRD_XDIM>>1); xdim >= 32; xdim >>= 1) {
        int tslot = threadIdx.x * smidim;
        //ccmCache will contain 3x32 (or length-size) (possibly equal) minimum scores 
        if(threadIdx.x < xdim &&
           qrypos + threadIdx.x + xdim < qrylen &&
           rfnpos + threadIdx.x + xdim < dbstrlen)
            StoreMinDstSrc(ccmCache + tslot, ccmCache + tslot + xdim * smidim);

        __syncthreads();
    }

    //unroll warp
    if(threadIdx.x < 32/*warpSize*/) {
        for(int xdim = (32>>1); xdim >= 1; xdim >>= 1) {
            int tslot = threadIdx.x * smidim;
            if(threadIdx.x < xdim)
                StoreMinDstSrc(ccmCache + tslot, ccmCache + tslot + xdim * smidim);
            __syncwarp();
        }
    }

    //write to gmem the minimum score that ensures at least 3 aligned positions:
    if(threadIdx.x == 2) {
        float d0 = GetD0(qrylenorg, dbstrlenorg);
        float d02s = GetD02s(d0);
        if(READCNST == READCNST_CALC2) d02s += D02s_PROC_INC;

        float min3 = ccmCache[threadIdx.x];

        //TODO: move the clause (maxnalnposs <= 3) along with the write to gmem up
        if(CP_LARGEDST_cmp < min3 || min3 < d02s ||
           GetGplAlnLength(qrylen, dbstrlen, qrypos, rfnpos) <= 3)
            //max number of alignment positions (maxnalnposs) <3;
            //use the dfault threshold
            min3 = d02s;
        else {//round the 3rd minimum score to the next multiple of 0.5:
            //obtained from d02s + k*0.5 >= min3
            min3 = d02s + ceilf((min3 - d02s) * 2.0f) * 0.5f;
            //d0 = floorf(min3);
            //d02s = min3 - d0;
            //if(d02s) min3 = d0 + ((d02s <= 0.5f)? 0.5f: 1.0f);
        }

        uint mloc = ((qryndx * maxnsteps + sfragfct) * nTAuxWorkingMemoryVars + tawmvLastD02s) * ndbCstrs;
        wrkmemaux[mloc + blockIdx.x/*dbstrndx*/] = min3;
    }
}

// =========================================================================
// Instantiations
//
#define INSTANTIATE_FindD02ThresholdsCCM_DPscan(tpREADCNST) \
    template \
    __global__ void FindD02ThresholdsCCM_DPscan<tpREADCNST>( \
        const uint ndbCstrs, const uint ndbCposs, const uint maxnsteps, \
        const float* __restrict__ tmpdpdiagbuffers, \
        const float* __restrict__ wrkmem, \
        float* __restrict__ wrkmemaux);

INSTANTIATE_FindD02ThresholdsCCM_DPscan(READCNST_CALC);
INSTANTIATE_FindD02ThresholdsCCM_DPscan(READCNST_CALC2);

// -------------------------------------------------------------------------

// -------------------------------------------------------------------------
// CalcCCMatrices64_DPscanExtended: calculate cross-covariance matrix 
// between the query and reference structures based on aligned positions 
// within given distance;
// Version for alignments obtained as a result of the exhaustive application of DP;
// NOTE: thread block is 1D and processes alignment fragment along structure
// positions;
// NOTE: Version for CUS1_TBINITSP_CCMCALC_XDIM==64!
// nqystrs, total number of queries;
// ndbCstrs, total number of reference structures in the chunk;
// ndbCposs, total number of db structure positions in the chunk;
// maxnsteps, max number of steps (blockIdx.z) to perform for each 
// reference structure;
// NOTE: memory pointers should be aligned!
// tmpdpalnpossbuffer, global address of the buffers of aligned positions;
// tmpdpdiagbuffers, temporary diagonal buffers filled with positional 
// scores;
// wrkmemaux, auxiliary working memory;
// wrkmem, working memory, including the section of CC data;
// 
template<int READCNST>
__global__
void CalcCCMatrices64_DPscanExtended(
    const uint nqystrs,
    const uint ndbCstrs,
    const uint ndbCposs,
    const uint dbxpad,
    const uint maxnsteps,
    const float* __restrict__ tmpdpalnpossbuffer,
    const float* __restrict__ tmpdpdiagbuffers,
    const float* __restrict__ wrkmemaux,
    float* __restrict__ wrkmem)
{
    // blockIdx.x is the block index of positions for query-reference pair;
    // blockIdx.y is the reference serial number;
    // blockIdx.z is the query serial number TIMES fragment factor;
    //cache for the cross-covarinace matrix and related data: 
    //no bank conflicts as long as inner-most dim is odd
    constexpr int neffds = twmvEndOfCCDataExt;//effective number of fields
    constexpr int smidim = neffds+1;
    __shared__ float ccmCache[smidim * CUS1_TBINITSP_CCMCALC_XDIM];
    //relative position index:
    const uint ndx0 = blockIdx.x * blockDim.x * CUS1_TBINITSP_CCMCALC_XFCT;
    const uint ndx = ndx0 + threadIdx.x;
    uint sfragfct = blockIdx.z / nqystrs;//fragment factor
    uint qryndx = blockIdx.z - sfragfct * nqystrs;//query serial number
    int qrylen, dbstrlen;//pseudo query and reference length, #matched positions
    //distances in positions to the beginnings of the query and reference structures:
    uint /*qrydst, */dbstrdst;
    constexpr int qrypos = 0;
    constexpr int rfnpos = 0;


    if(threadIdx.x == 0) {
        //NOTE: reuse ccmCache to read convergence flag at sfragfct==0:
        uint mloc = ((qryndx * maxnsteps + 0/*sfragfct*/) * nTAuxWorkingMemoryVars + tawmvConverged) * ndbCstrs;
        ccmCache[6] = wrkmemaux[mloc + blockIdx.y/*dbstrndx*/];
    }

    __syncthreads();

    if(ccmCache[6])
        //(NOTE:any type of convergence applies);
        //all threads in the block exit;
        return;

    //NOTE: no sync as long as ccmCache cell for convergence is not overwritten;


    //reuse ccmCache
    if(threadIdx.x == 0) {
        ((int*)ccmCache)[1] = GetDbStrDst(blockIdx.y);
        //((int*)ccmCache)[3] = GetQueryDst(qryndx);
    }
    if(threadIdx.x == tawmvNAlnPoss + 32) {
        //NOTE: reuse ccmCache to read #matched positions (tawmvNAlnPoss) written at sfragfct;
        //NOTE: use different warp; structure-specific-formatted data;
        uint mloc = ((qryndx * maxnsteps + sfragfct) * nTAuxWorkingMemoryVars) * ndbCstrs;
        ccmCache[threadIdx.x] = wrkmemaux[mloc + (threadIdx.x-32) * ndbCstrs + blockIdx.y/*dbstrndx*/];
    }

    __syncthreads();


    //NOTE: no bank conflict when two threads from the same warp access the same address;
    //blockDim.x includes several warps
    dbstrdst = ((int*)ccmCache)[1];
    //qrydst = ((int*)ccmCache)[3];
    qrylen = dbstrlen = ccmCache[tawmvNAlnPoss+32];

    __syncthreads();


    if(qrylen <= qrypos + ndx0 || dbstrlen <= rfnpos + ndx0)
        //all threads in the block exit if thread 0 is out of bounds
        return;


    InitCCMCacheExtended<smidim,6,neffds>(ccmCache);

    if(threadIdx.x == 0) {
        //NOTE: reuse ccmCache[0] to contain twmvLastD02s, ccmCache[1] twmvNalnposs
        //structure-specific-formatted data
        uint mloc = ((qryndx * maxnsteps + sfragfct) * nTAuxWorkingMemoryVars + tawmvLastD02s) * ndbCstrs;
        ccmCache[0] = wrkmemaux[mloc + blockIdx.y/*dbstrndx*/];

        if(READCNST == READCNST_CALC2) {
            mloc = ((qryndx * maxnsteps + sfragfct) * ndbCstrs + blockIdx.y/*dbstrndx*/) * nTWorkingMemoryVars;
            ccmCache[1] = wrkmem[mloc + twmvNalnposs];
        }
    }

    __syncthreads();

    float d02s = ccmCache[0];

    if(READCNST == READCNST_CALC2) {
        int nalnposs = ccmCache[1];
        if(nalnposs == dbstrlen)
            //dbstrlen is #originally aligned positions;
            //all threads in the block exit;
            return;
    }

    //cache will be overwritten below, sync
    __syncthreads();


    //cache initialization divided into two parts for a more efficient use of registers
    InitCCMCacheExtended<smidim,0,6>(ccmCache);

    const int dblen = ndbCposs + dbxpad;
    //offset to the beginning of the data along the y axis 
    // wrt query qryndx and maxnsteps: 
    const int yofff = (qryndx * maxnsteps + sfragfct) * dblen * nTDPAlignedPoss;

    for(int i = 0; i < CUS1_TBINITSP_CCMCALC_XFCT; i++) {
        //manually unroll along data blocks by a factor of CUS1_TBINITSP_CCMCALC_XFCT
        int mloc = (qryndx * maxnsteps + sfragfct) * ndbCposs;
        int pos0 = ndx + i * blockDim.x;//position index starting from 0
        if(!(qrypos + pos0 < qrylen && rfnpos + pos0 < dbstrlen))
            break;
        //starting position in tmpdpalnpossbuffer for a pair:
        //NOTE: aligned coordinates in tmpdpalnpossbuffer are in the reverse order!
        //NOTE: qrypos == rfnpos as well as qrylen == dbstrlen here
        int dppos = yofff + dbstrdst + dbstrlen-1 - (rfnpos + pos0);
        UpdateCCMOneAlnPos_DPExtended<smidim>(//no sync;
            d02s,
            dppos, dblen,
            mloc + dbstrdst + pos0,//position for scores
            tmpdpalnpossbuffer,//coordinates
            tmpdpdiagbuffers,//scores
            ccmCache//reduction output
        );
    }

    //sync now:
    __syncthreads();

    //unroll by a factor 2
    if(threadIdx.x < (CUS1_TBINITSP_CCMCALC_XDIM>>1)) {
        #pragma unroll
        for(int i = 0; i < neffds; i++)
            ccmCache[threadIdx.x * smidim + i] +=
                ccmCache[(threadIdx.x + (CUS1_TBINITSP_CCMCALC_XDIM>>1)) * smidim + i];
    }

    __syncthreads();

    //unroll warp
    if(threadIdx.x < 32) {
        #pragma unroll
        for(int i = 0; i < neffds; i++) {
            float sum = ccmCache[threadIdx.x * smidim + i];
            sum = mywarpreducesum(sum);
            //write to the first data slot of SMEM
            if(threadIdx.x == 0) ccmCache[i] = sum;
        }
    }

    //in case of neffds gets larger than warpSize
    __syncthreads();

    //add the result and write to global memory
    if(threadIdx.x < neffds) {
        uint mloc = ((qryndx * maxnsteps + sfragfct) * ndbCstrs + blockIdx.y/*dbstrndx*/) * nTWorkingMemoryVars;
        atomicAdd(&wrkmem[mloc + threadIdx.x], ccmCache[threadIdx.x]);
    }
}

// =========================================================================
// Instantiations
//
#define INSTANTIATE_CalcCCMatrices64_DPscanExtended(tpREADCNST) \
    template \
    __global__ void CalcCCMatrices64_DPscanExtended<tpREADCNST>( \
        const uint nqystrs, const uint ndbCstrs, const uint ndbCposs, const uint dbxpad, \
        const uint maxnsteps, \
        const float* __restrict__ tmpdpalnpossbuffer, \
        const float* __restrict__ tmpdpdiagbuffers, \
        const float* __restrict__ wrkmemaux, \
        float* __restrict__ wrkmem);

INSTANTIATE_CalcCCMatrices64_DPscanExtended(READCNST_CALC);
INSTANTIATE_CalcCCMatrices64_DPscanExtended(READCNST_CALC2);

// -------------------------------------------------------------------------



// -------------------------------------------------------------------------
// CalcScoresUnrl_DPscan: calculate/reduce scores for obtained 
// superpositions; version for alignments obtained by exhaustively applying 
// DP; 
// NOTE: save partial sums;
// NOTE: thread block is 1D and processes alignment fragment along structure
// positions;
// NOTE: Universal version for any CUS1_TBSP_SCORE_XDIM multiple of 32;
// SAVEPOS, template parameter to request saving positional scores;
// CHCKALNLEN, template parameter for checking whether alignment length has 
// changed;
// nqystrs, total number of queries;
// ndbCstrs, total number of reference structures in the chunk;
// ndbCposs, total number of db structure positions in the chunk;
// maxnsteps, max number of steps (blockIdx.z) to perform for each 
// reference structure;
// NOTE: memory pointers should be aligned!
// tmpdpalnpossbuffer, coordinates of matched positions obtained by DP;
// wrkmemtm, working memory of transformation matrices;
// wrkmem, working memory for cross-covariance data;
// wrkmemaux, auxiliary working memory;
// tmpdpdiagbuffers, temporary diagonal buffers reused here for saving 
// positional scores;
// NOTE: keep #registers <= 32
// 
template<int SAVEPOS, int CHCKALNLEN>
__global__
void CalcScoresUnrl_DPscan(
    const uint nqystrs,
    const uint ndbCstrs,
    const uint ndbCposs,
    const uint dbxpad,
    const uint maxnsteps,
    const float* __restrict__ tmpdpalnpossbuffer,
    const float* __restrict__ wrkmemtm,
    const float* __restrict__ wrkmem,
    float* __restrict__ wrkmemaux,
    float* __restrict__ tmpdpdiagbuffers)
{
    // blockIdx.x is the block index of positions for query-reference pair;
    // blockIdx.y is the reference serial number;
    // blockIdx.z is the query serial number TIMES fragment factor;
    //no bank conflicts as long as inner-most dim is odd
    constexpr int pad = 1;//padding
    //cache for scores and transformation matrix: 
    __shared__ float scvCache[pad + CUS1_TBSP_SCORE_XDIM + nTTranformMatrix];
    //pointer to transformation matrix;
    float* tfmCache = scvCache + pad + CUS1_TBSP_SCORE_XDIM;
    //relative position index:
    const uint ndx0 = blockIdx.x * blockDim.x * CUS1_TBSP_SCORE_XFCT;
    const uint ndx = ndx0 + threadIdx.x;
    uint sfragfct = blockIdx.z / nqystrs;//fragment factor
    uint qryndx = blockIdx.z - sfragfct * nqystrs;//query serial number
    int qrylenorg, dbstrlenorg;//original query and reference lengths
    int qrylen, dbstrlen;//pseudo query and reference length, #matched positions
    //distances in positions to the beginnings of the query and reference structures:
    uint /*qrydst, */dbstrdst;
    constexpr int qrypos = 0;
    constexpr int rfnpos = 0;


    if(threadIdx.x == 0) {
        //NOTE: reuse ccmCache to read convergence flag at sfragfct==0:
        uint mloc = ((qryndx * maxnsteps + 0/*sfragfct*/) * nTAuxWorkingMemoryVars + tawmvConverged) * ndbCstrs;
        scvCache[6] = wrkmemaux[mloc + blockIdx.y/*dbstrndx*/];
    }

    __syncthreads();

    if(scvCache[6])
        //(NOTE:any type of convergence applies);
        //all threads in the block exit;
        return;

    //NOTE: no sync as long as scvCache cell for convergence is not overwritten;


    //NOTE: pps2DLen and pps2DDist assumed to be adjacent: see PM2DVectorFields.h!
    //reuse scvCache
    if(threadIdx.x < 2) {
        GetDbStrLenDst(blockIdx.y, (int*)scvCache);
        GetQueryLenDst(qryndx, (int*)scvCache + 2);
    }
    if(threadIdx.x == tawmvNAlnPoss + 32) {
        //NOTE: reuse ccmCache to read #matched positions (tawmvNAlnPoss) written at sfragfct;
        //NOTE: use different warp; structure-specific-formatted data;
        uint mloc = ((qryndx * maxnsteps + sfragfct) * nTAuxWorkingMemoryVars) * ndbCstrs;
        scvCache[threadIdx.x] = wrkmemaux[mloc + (threadIdx.x-32) * ndbCstrs + blockIdx.y/*dbstrndx*/];
    }

    __syncthreads();

    //NOTE: no bank conflict when two threads from the same warp access the same address;
    //blockDim.x includes several warps
    dbstrlenorg = ((int*)scvCache)[0]; dbstrdst = ((int*)scvCache)[1];
    qrylenorg = ((int*)scvCache)[2]; //qrydst = ((int*)scvCache)[3];
    qrylen = dbstrlen = scvCache[tawmvNAlnPoss+32];

    __syncthreads();


    if(qrylen <= qrypos + ndx0 || dbstrlen <= rfnpos + ndx0)
        //all threads in the block exit if thread 0 is out of bounds
        return;


    if(CHCKALNLEN == CHCKALNLEN_CHECK) {
        if(threadIdx.x == 0) {
            //NOTE: reuse scvCache[0] for twmvNalnposs
            uint mloc = ((qryndx * maxnsteps + sfragfct) * ndbCstrs + blockIdx.y/*dbstrndx*/) * nTWorkingMemoryVars;
            scvCache[0] = wrkmem[mloc + twmvNalnposs];
        }

        __syncthreads();

        int nalnposs = scvCache[0];
        if(nalnposs == dbstrlen)
            //dbstrlen is #originally aligned positions;
            //score has been calculated before; 
            //all threads in the block exit;
            return;
        //no sync as scvCache[0(pad-1)] is not used below
    }


    //threshold calculated for the original lengths
    float d02 = GetD02(qrylenorg, dbstrlenorg);

    //initialize cache
    scvCache[pad + threadIdx.x] = 0.0f;

    //read transformation matrix for query-reference pair
    if(threadIdx.x < nTTranformMatrix) {
        uint mloc = ((qryndx * maxnsteps + sfragfct) * ndbCstrs + blockIdx.y/*dbstrndx*/) * nTTranformMatrix;
        tfmCache[threadIdx.x] = wrkmemtm[mloc + threadIdx.x];
    }

    __syncthreads();


    const int dblen = ndbCposs + dbxpad;
    //offset to the beginning of the data along the y axis 
    // wrt query qryndx and maxnsteps: 
    const int yofff = (qryndx * maxnsteps + sfragfct) * dblen * nTDPAlignedPoss;

    #pragma unroll
    for(int i = 0; i < CUS1_TBSP_SCORE_XFCT; i++) {
        //manually unroll along data blocks by a factor of CUS1_TBSP_SCORE_XFCT
        int mloc = (qryndx * maxnsteps + sfragfct) * ndbCposs;
        int pos0 = ndx + i * blockDim.x;//position index starting from 0
        if(!(qrypos + pos0 < qrylen && rfnpos + pos0 < dbstrlen))
            break;
        //starting position in tmpdpalnpossbuffer for a pair:
        //NOTE: aligned coordinates in tmpdpalnpossbuffer are in the reverse order!
        //NOTE: qrypos == rfnpos as well as qrylen == dbstrlen here
        int dppos = yofff + dbstrdst + dbstrlen-1 - (rfnpos + pos0);
        UpdateOneAlnPosScore_DPRefined<SAVEPOS,CHCKDST_NOCHECK>(//no sync;
            d02, d02,
            dppos, dblen,
            mloc + dbstrdst + pos0,//position for scores
            tmpdpalnpossbuffer,//coordinates
            tfmCache,//tfm. mtx.
            scvCache + pad,//score cache
            tmpdpdiagbuffers//scores written to gmem
        );
    }

    //sync now:
    __syncthreads();

    //unroll until reaching warpSize 
    #pragma unroll
    for(int xdim = (CUS1_TBSP_SCORE_XDIM>>1); xdim >= 32; xdim >>= 1) {
        if(threadIdx.x < xdim)
            scvCache[pad + threadIdx.x] +=
                scvCache[pad + threadIdx.x + xdim];

        __syncthreads();
    }

    //unroll warp
    if(threadIdx.x < 32/*warpSize*/) {
        float sum = scvCache[pad + threadIdx.x];
        sum = mywarpreducesum(sum);
        //write to the first data slot of SMEM
        if(threadIdx.x == 0) scvCache[0] = sum;
    }

    //add the score and write to global memory
    if(threadIdx.x == 0) {
        //structure-specific-formatted data; scvCache[0] is the reduced score
        uint mloc = ((qryndx * maxnsteps + sfragfct) * nTAuxWorkingMemoryVars + tawmvScore) * ndbCstrs;
        atomicAdd(&wrkmemaux[mloc + blockIdx.y/*dbstrndx*/], scvCache[0]);
    }
}

// -------------------------------------------------------------------------
// Instantiations
// 
#define INSTANTIATE_CalcScoresUnrl_DPscan(tpSAVEPOS,tpCHCKALNLEN) \
    template __global__ void CalcScoresUnrl_DPscan<tpSAVEPOS,tpCHCKALNLEN>( \
        const uint nqystrs, const uint ndbCstrs, const uint ndbCposs, const uint dbxpad, \
        const uint maxnsteps, \
        const float* __restrict__ tmpdpalnpossbuffer, \
        const float* __restrict__ wrkmemtm, \
        const float* __restrict__ wrkmem, \
        float* __restrict__ wrkmemaux, \
        float* __restrict__ tmpdpdiagbuffers);

INSTANTIATE_CalcScoresUnrl_DPscan(SAVEPOS_SAVE,CHCKALNLEN_NOCHECK);
INSTANTIATE_CalcScoresUnrl_DPscan(SAVEPOS_SAVE,CHCKALNLEN_CHECK);
INSTANTIATE_CalcScoresUnrl_DPscan(SAVEPOS_NOSAVE,CHCKALNLEN_CHECK);

// -------------------------------------------------------------------------
