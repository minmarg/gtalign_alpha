#include "hip/hip_runtime.h"
/***************************************************************************
 *   Copyright (C) 2021-2023 Mindaugas Margelevicius                       *
 *   Institute of Biotechnology, Vilnius University                        *
 ***************************************************************************/

#include "libutil/cnsts.h"
#include "libutil/macros.h"
#include "libgenp/gproc/gproc.h"
#include "libgenp/gdats/PM2DVectorFields.h"

#include "libmycu/cucom/cucommon.h"
#include "libmycu/cucom/warpscan.cuh"
#include "libmycu/cuproc/cuprocconf.h"
#include "libmycu/culayout/cuconstant.cuh"
#include "libmycu/custages/stagecnsts.cuh"
#include "libmycu/custages/covariance.cuh"
#include "libmycu/custages/covariance_plus.cuh"
#include "libmycu/custages/covariance_refn.cuh"
#include "libmycu/custages/covariance_dp_refn.cuh"
#include "libmycu/custages2/covariance_complete.cuh"
#include "libmycu/custages2/covariance_refn_complete.cuh"
#include "covariance_dp_refn_complete.cuh"

// =========================================================================
// FragmentBasedDPAlignmentRefinement: refine alignment and its boundaries 
// within the single kernel's actions to obtain favorable superposition;
// WRITEFRAGINFO, template parameter, flag of whether refined fragment 
// boundaries should be saved;
// CONDITIONAL, template parameter, flag of writing the score if it's 
// greater at the same location;
// TFM_DINV, use doubly inverted transformation matrices under suitable conditions;
// readlocalconv, flag of reading local convergence flag;
// nmaxconvit, maximum number of superposition iterations;
// nqystrs, total number of query structures in the chunk;
// ndbCstrs, total number of reference structures in the chunk;
// ndbCposs, total number of reference positions in the chunk;
// dbxpad, #pad positions along the dimension of reference structures;
// nmaxsubfrags, total number of fragment lengths to consider;
// maxnsteps, total number of steps that should be performed for each reference structure;
// sfragstep, step size to traverse subfragments;
// maxalnmax, maximum alignment length across all query-reference pairs;
// tmpdpalnpossbuffer, coordinates of matched positions obtained by DP;
// tmpdpdiagbuffers, temporary diagonal buffers for positional scores;
// wrkmemtmibest, working memory for best-performing transformation matrices;
// wrkmemaux, auxiliary working memory (includes the section of scores);
// 
template<bool WRITEFRAGINFO, bool CONDITIONAL, bool TFM_DINV>
__global__ 
void FragmentBasedDPAlignmentRefinement(
    const bool readlocalconv,
    const int nmaxconvit,
//     const uint nqystrs,
    const uint ndbCstrs,
    const uint ndbCposs,
    const uint dbxpad,
    const uint nmaxsubfrags,
    const uint maxnsteps,
    const int sfragstep,
    const int /*maxalnmax*/,
    const float* __restrict__ tmpdpalnpossbuffer,
    float* __restrict__ tmpdpdiagbuffers,
    float* __restrict__ wrkmemtmibest,
    float* __restrict__ wrkmemaux)
{
    uint dbstrndx = blockIdx.x;//reference serial number
    uint sfragfctxndx = blockIdx.y;//fragment factor x fragment length index
    uint sfragfct = sfragfctxndx / nmaxsubfrags;//fragment factor
    uint sfragndx = sfragfctxndx - sfragfct * nmaxsubfrags;//fragment length index
    uint qryndx = blockIdx.z;//query serial number
    //cache for the cross-covarinace matrix and related data: 
    //no bank conflicts as long as inner-most dim is odd
    enum {neffds = twmvEndOfCCDataExt,//effective number of fields
        smidim = neffds+1};
    __shared__ float ccmCache[
        smidim * CUS1_TBINITSP_COMPLETEREFINE_XDIM + twmvEndOfCCDataExt * 2 + nTTranformMatrix];
//     __shared__ float ccmLast[twmvEndOfCCDataExt];
//     __shared__ float tfmCache[twmvEndOfCCDataExt];//twmvEndOfCCDataExt>nTTranformMatrix
    float* ccmLast = ccmCache + smidim * CUS1_TBINITSP_COMPLETEREFINE_XDIM;
    float* tfmCache = ccmLast + twmvEndOfCCDataExt;//twmvEndOfCCDataExt>nTTranformMatrix
    float* tfmBest = tfmCache + twmvEndOfCCDataExt;//of size nTTranformMatrix

    int qrylenorg, dbstrlenorg;//original query and reference lengths
    int qrylen, dbstrlen;//pseudo query and reference length, #matched positions
    //distances in positions to the beginnings of the query and reference structures:
    uint /*qrydst, */dbstrdst;
    enum {qrypos = 0, rfnpos = 0};
    int sfragpos, fraglen;


///     //get sfragfct and sfragndx given sfragfctxndx
///     GetSubfragFctAndNdx(sfragfct, sfragndx,
///         sfragfctxndx, nmaxsubfrags, sfragstep, maxalnmax);
/// 
///     //all threads exits: out of bounds
///     if(nmaxsubfrags <= sfragndx) return;


    if(threadIdx.x == 0) {
        uint mloc0 = ((qryndx * maxnsteps + 0) * nTAuxWorkingMemoryVars + tawmvConverged) * ndbCstrs;
        uint mloc = ((qryndx * maxnsteps + sfragfctxndx) * nTAuxWorkingMemoryVars + tawmvConverged) * ndbCstrs;
        ccmCache[6] = ccmCache[7] = wrkmemaux[mloc0 + dbstrndx];
        if(readlocalconv && sfragfctxndx != 0) ccmCache[7] = wrkmemaux[mloc + dbstrndx];
    }

    __syncthreads();

    if((((int)(ccmCache[6])) & (CONVERGED_LOWTMSC_bitval)) || ccmCache[7])
        //(NOTE:any type of convergence applies locally and CONVERGED_LOWTMSC_bitval globally);
        //all threads in the block exit;
        return;

    //NOTE: no sync as long ccmCache cell for convergence is not overwritten;

    //NOTE: pps2DLen and pps2DDist assumed to be adjacent: see PM2DVectorFields.h!
    //reuse ccmCache
    if(threadIdx.x < 2) {
        GetDbStrLenDst(dbstrndx, (int*)ccmCache);
        //GetQueryLenDst(qryndx, (int*)ccmCache + 2);
        if(threadIdx.x == 0) ((int*)ccmCache)[2] = GetQueryLength(qryndx);
    }

    //NOTE: use a different warp for structure-specific-formatted data;
#if (CUS1_TBINITSP_COMPLETEREFINE_XDIM >= 64)
    if(threadIdx.x == tawmvNAlnPoss + 32) {
#else
    if(threadIdx.x == tawmvNAlnPoss) {
#endif
        //NOTE: reuse ccmCache to read #matched positions;
        //NOTE: tawmvNAlnPoss written at sfragfct==0:
        uint mloc = ((qryndx * maxnsteps + 0) * nTAuxWorkingMemoryVars) * ndbCstrs;
        ccmCache[tawmvNAlnPoss] = wrkmemaux[mloc + tawmvNAlnPoss * ndbCstrs + dbstrndx];
    }

    __syncthreads();


    dbstrdst = ((int*)ccmCache)[1];
    //qrydst = ((int*)ccmCache)[3];
    qrylen = dbstrlen = ccmCache[tawmvNAlnPoss];
    sfragpos = sfragfct * sfragstep;
    dbstrlenorg = ((int*)ccmCache)[0];
    qrylenorg = ((int*)ccmCache)[2];

    __syncthreads();


    fraglen = GetFragLength(qrylen, dbstrlen, qrypos, rfnpos, sfragndx);
    if(fraglen < 1) 
        //all threads in the block exit
        return;

    if(qrylen + sfragstep <= qrypos + sfragpos + fraglen ||
       dbstrlen + sfragstep <= rfnpos + sfragpos + fraglen)
        //all threads in the block exit
        return;


    //threshold calculated for the original lengths
    const float d0 = GetD0(qrylenorg, dbstrlenorg);
    const float d02 = SQRD(d0);
    const float d82 = GetD82(qrylenorg, dbstrlenorg);
    float dst32 = CP_LARGEDST;
    float best = 0.0f;//best score obtained

    CalcCCMatrices64_DPRefined_Complete<smidim,neffds>(
        qryndx,  ndbCposs, dbxpad,  maxnsteps, sfragfctxndx, dbstrdst, fraglen,
        qrylen, dbstrlen,  qrypos + sfragpos, rfnpos + sfragpos,
        tmpdpalnpossbuffer, ccmCache);


    for(int cit = 0; cit < nmaxconvit + 2; cit++)
    {
        if(0 < cit) {
            CalcCCMatrices64_DPRefinedExtended_Complete<smidim,neffds>(
                (cit < 2)? READCNST_CALC: READCNST_CALC2,
                qryndx, ndbCposs, dbxpad, maxnsteps, sfragfctxndx, dbstrdst,
                qrylen, dbstrlen, qrypos, rfnpos,  d0, dst32,
                tmpdpdiagbuffers, tmpdpalnpossbuffer, ccmCache);

            CheckConvergence64Refined_Complete(ccmCache, ccmLast);
            if(ccmLast[0]) break;//converged
            __syncthreads();//prevent overwriting ccmLast[0]
        }

        //NOTE: synced above and below before ccmCache gets updated;
        if(ccmCache[twmvNalnposs] < 1.0f) break;

        SaveCCMData_Complete(ccmCache, tfmCache, ccmLast);
        //NOTE: tfmCache updated by the first warp; 
        //NOTE: CalcTfmMatrices_Complete uses only the first warp;
        //NOTE: ccmLast not used until the first syncthreads below;
        __syncwarp();

        CalcTfmMatrices_Complete<TFM_DINV>(tfmCache, qrylenorg, dbstrlenorg);
        //all threads synced and see the tfm

        CalcScoresUnrl_DPRefined_Complete(
            (cit < 1)? READCNST_CALC: READCNST_CALC2,
            qryndx, ndbCposs, dbxpad, maxnsteps, sfragfctxndx, dbstrdst,
            qrylen, dbstrlen, qrypos, rfnpos,  d0, d02, d82,
            tmpdpdiagbuffers, tmpdpalnpossbuffer, tfmCache, ccmCache+1);

        //distance threshold for at least three aligned pairs:
        dst32 = ccmCache[2];

        //NOTE: no sync inside:
        SaveLocalBestScoreAndTM(best, ccmCache[1]/*score*/, tfmCache, tfmBest);

        //sync all threads to see dst32 (and prevent overwriting the cache):
        __syncthreads();
    }

    //NOTE: synced either after the last cit or convergence check:
    SaveBestScoreAndTM_Complete<WRITEFRAGINFO,CONDITIONAL>(
        best,  qryndx, dbstrndx, ndbCstrs, 
        maxnsteps, sfragfctxndx, sfragndx, sfragpos,
        tfmBest, wrkmemtmibest, wrkmemaux);
}

// -------------------------------------------------------------------------
// Instantiations
// 
#define INSTANTIATE_FragmentBasedDPAlignmentRefinement(tpWRITEFRAGINFO,tpCONDITIONAL,tpTFM_DINV) \
    template __global__ void FragmentBasedDPAlignmentRefinement<tpWRITEFRAGINFO,tpCONDITIONAL,tpTFM_DINV>( \
        const bool readlocalconv, \
        const int nmaxconvit, /*const uint nqystrs,*/ \
        const uint ndbCstrs, const uint ndbCposs, const uint dbxpad, \
        const uint nmaxsubfrags, const uint maxnsteps, \
        const int sfragstep, const int maxalnmax, \
        const float* __restrict__ tmpdpalnpossbuffer, \
        float* __restrict__ tmpdpdiagbuffers, \
        float* __restrict__ wrkmemtmibest, \
        float* __restrict__ wrkmemaux);

INSTANTIATE_FragmentBasedDPAlignmentRefinement(false/* true */,true,false);
INSTANTIATE_FragmentBasedDPAlignmentRefinement(false/* true */,false,false);

INSTANTIATE_FragmentBasedDPAlignmentRefinement(false/* true */,true,true);
INSTANTIATE_FragmentBasedDPAlignmentRefinement(false/* true */,false,true);

// =========================================================================
